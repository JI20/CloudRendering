#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAStream.h>
#include "perPixelKernel.hpp"

#define MAX_CHANNELS (4)


inline int iceil(int x, int y) {
    return 1 + ((x - 1) / y);
}

__global__ void perPixelKernelForwardKernel(
        float* image,
        float* weights,
        float* output,
        const int kernelSize, const int h, const int w, const int numChannels,
        const int numBatches
        ) {

    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int batch = threadIdx.z + blockIdx.z * blockDim.z;
    //const int channel = threadIdx.z + blockIdx.z * blockDim.z;
    //float result = 0;

    float result[MAX_CHANNELS];
    for (int c = 0; c < numChannels; c++){
        result[c] = 0;
    }

    if (x >= w || y >= h) {
        return;
    }
    int weightIdx = 0;
    for (int dy = -kernelSize/2; dy <= kernelSize/2; dy++){
        int ky = y + dy;
        for (int dx = -kernelSize/2; dx <= kernelSize/2; dx++){
            int kx = x + dx;
            if (kx >= 0 && kx < w && ky >= 0 && ky < h) {
                float weight = weights[batch * w * h * kernelSize * kernelSize + weightIdx * w * h + y * w + x];
                for (int c = 0; c < numChannels; c++) {
                    result[c] += weight * image[batch * w * h * numChannels + c * w * h + ky * w + kx];
                }
            }
            weightIdx++;
        }
    }
    for (int c = 0; c < numChannels; c++){
        output[batch * w * h * numChannels+ c * w * h + y * w + x] = result[c];
    }
}

torch::Tensor perPixelKernelCuda(torch::Tensor image, torch::Tensor weights, int64_t kernelSize){
    if (image.sizes().size() != 4) {
        throw std::runtime_error("Error in perPixelKernelCuda: image.sizes().size() != 4.");
    }
    if (weights.sizes().size() != 4) {
        throw std::runtime_error("Error in perPixelKernelCuda: weights.sizes().size() != 4.");
    }

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const int64_t N = image.size(0);
    const int64_t C = image.size(1);
    const int64_t H = image.size(2);
    const int64_t W = image.size(3);

    if (H != weights.size(2) || W != weights.size((3))) {
        throw std::runtime_error("Error, image and weight sizes dont match");
    }
    if (C > MAX_CHANNELS) {
        throw std::runtime_error("Error, must have <= 4 channels");
    }

    torch::Tensor result = torch::zeros_like(image);
    auto imageAccessor = image.packed_accessor32<float, 4>();
    auto weightsAccessor = weights.packed_accessor32<float, 4>();
    auto resultAccessor = result.packed_accessor32<float, 4>();

    dim3 blockDim(16, 16, 1);
    dim3 gridDim(iceil(W, blockDim.x), iceil(H, blockDim.y), N);

    perPixelKernelForwardKernel<<<gridDim, blockDim, 0, stream>>> (
            (float*)image.data_ptr(), (float*)weights.data_ptr(), (float*)result.data_ptr(),
            kernelSize, H, W, C, N
    );

    return result;
}