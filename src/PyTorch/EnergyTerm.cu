#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 *
 * Copyright (c) 2024, Christoph Neuhauser
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 *
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <iostream>
#include <Math/Math.hpp>
#include "EnergyTerm.cuh"

#define F_PI 3.1415926535897932f
#define F_TWO_PI (2.0f * F_PI)

__device__ float sign(float x) {
    return x > 0.0f ? 1.0f : (x < 0.0f ? -1.0f : 0.0f);
}

__device__ int iclamp(int x, int l, int u) {
    return x <= l ? l : (x >= u ? u : x);
}

__global__ void updateObservationFrequencyFieldsKernel(
        uint32_t depth, uint32_t height, uint32_t width, uint32_t numBinsX, uint32_t numBinsY, uint32_t numBins,
        float camPosX, float camPosY, float camPosZ, const float* __restrict__ transmittanceField,
        float* __restrict__ obsFreqField, float* __restrict__ angularObsFreqField) {
    uint32_t idX = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idY = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t idZ = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t linearIdx = idX + (idY + idZ * height) * width;
    if (idX >= width || idY >= height || idZ >= depth) {
        return;
    }
    float transmittance = transmittanceField[linearIdx];

    float dx = float(idX) - camPosX;
    float dy = float(idY) - camPosY;
    float dz = float(idZ) - camPosZ;
    float len = sqrtf(dx * dx + dy * dy + dz * dz);
    dx /= len;
    dy /= len;
    dz /= len;
    float theta = acosf(dz);
    float phi = sign(dy) * acosf(dx / sqrt(dx * dx + dy * dy));
    auto binIdxX = uint32_t(iclamp(int(fmodf(theta + F_TWO_PI, F_PI) / F_PI * float(numBinsX)), 0, int(numBinsX) - 1));
    auto binIdxY = uint32_t(iclamp(int(fmodf(phi + F_TWO_PI, F_TWO_PI) / F_TWO_PI * float(numBinsY)), 0, int(numBinsY) - 1));
    obsFreqField[linearIdx] += transmittance;
    if (linearIdx * numBins + binIdxX + binIdxY * numBinsX < 1) {
        angularObsFreqField[linearIdx * numBins + binIdxX + binIdxY * numBinsX] += transmittance;
    }
}

__global__ void computeEnergyKernel(
        uint32_t depth, uint32_t height, uint32_t width, uint32_t numBinsX, uint32_t numBinsY, uint32_t numBins,
        uint32_t numCams, float gamma,
        const float* __restrict__ obsFreqField, const float* __restrict__ angularObsFreqField,
        const uint8_t* __restrict__ occupancyField, float* __restrict__ energyTermField) {
    uint32_t idX = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idY = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t idZ = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t linearIdx = idX + (idY + idZ * height) * width;
    if (idX >= width || idY >= height || idZ >= depth) {
        return;
    }

    bool isOccupied = occupancyField[linearIdx] > 0u;
    if (isOccupied) {
        float energyTermLocal = 0.0f;
        const float fN = 1.0f / float(numCams);
        energyTermLocal += pow(obsFreqField[linearIdx] * fN, gamma);

        // Use total variation distance.
        float entrySum = 0.0f, TV = 0.0f;
        for (int binIdx = 0; binIdx < numBins; binIdx++) {
            entrySum += angularObsFreqField[linearIdx * numBins + binIdx];
        }
        if (entrySum > 1e-6f) {
            float invEntrySum = 1.0f / entrySum;
            for (int binIdx = 0; binIdx < numBins; binIdx++) {
                TV += abs(angularObsFreqField[linearIdx * numBins + binIdx] * invEntrySum - invEntrySum);
            }
            energyTermLocal += 1.0f - TV * 0.5f;
        }
        energyTermField[linearIdx] = energyTermLocal;
    }
}

void updateObservationFrequencyFieldsImpl(
        hipStream_t stream, uint32_t depth, uint32_t height, uint32_t width, uint32_t numBinsX, uint32_t numBinsY,
        const glm::vec3& camPos, const float* transmittanceField, float* obsFreqField, float* angularObsFreqField) {
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(sgl::uiceil(width, blockDim.x), sgl::uiceil(height, blockDim.y), sgl::uiceil(depth, blockDim.z));
    updateObservationFrequencyFieldsKernel<<<gridDim, blockDim, 0, stream>>> (
            depth, height, width, numBinsX, numBinsY, numBinsX * numBinsY, camPos.x, camPos.y, camPos.z,
            transmittanceField, obsFreqField, angularObsFreqField);
}

void computeEnergyImpl(
        hipStream_t stream, uint32_t depth, uint32_t height, uint32_t width, uint32_t numBinsX, uint32_t numBinsY,
        uint32_t numCams, float gamma,
        const float* obsFreqField, const float* angularObsFreqField,
        const uint8_t* occupancyField, float* energyTermField) {
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(sgl::uiceil(width, blockDim.x), sgl::uiceil(height, blockDim.y), sgl::uiceil(depth, blockDim.z));
    computeEnergyKernel<<<gridDim, blockDim, 0, stream>>> (
            depth, height, width, numBinsX, numBinsY, numBinsX * numBinsY, numCams, gamma,
            obsFreqField, angularObsFreqField, occupancyField, energyTermField);
}
